#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <cassert>
#include <cmath>
#include <cstring>
#include "vector.cuh"

const int THREADS = 1024;

int main() {
	std::mt19937 rnd(0x114514);

	std::size_t n = 1 << 10;
	std::size_t dim = 1 << 15;
	float* a = (float*)malloc(n * dim * sizeof(float));
	for(int i = 0; i < n * dim; ++i) {
		a[i] = rnd() / 1e5;
		//a[i] = 1;
	}
	float* b = (float*)malloc(dim * sizeof(float));
	float* c = (float*)malloc(dim * sizeof(float));
	std::memset(b, 0, dim * sizeof(float));

	float* device_a = NULL, *device_b = NULL;
	assert(hipMalloc((void**) &device_a, n * dim * sizeof(float)) == hipSuccess);
	assert(hipMalloc((void**) &device_b, dim * sizeof(float)) == hipSuccess);
	assert(hipMemcpy(device_a, a, n * dim * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
	assert(hipMemset(device_b, 0, dim * sizeof(float)) == hipSuccess);

	for(int t = 0; t < (1 << 10); ++t) {
		Accumulate<<<dim / THREADS + 1, THREADS>>>(device_a, n, dim, device_b);
	}

	for(int i = 0; i < n; ++i) {
		for(int j = 0; j < dim; ++j) {
			b[j] += a[i * dim + j];
		}
	}

	assert(hipMemcpy(c, device_b, dim * sizeof(float), hipMemcpyDeviceToHost) == hipSuccess);

	for(int i = 0; i < dim; ++i) assert(std::fabs(c[i] - b[i]) / std::max(fabs(b[i]), 1.0f) < 1e-3);

	assert(hipFree(device_a) == hipSuccess);
	assert(hipFree(device_b) == hipSuccess);
	free(a);
	free(b);

	return 0;
}
