#include "hip/hip_runtime.h"
#include "vector.cuh"
#include <cassert>
#include <cstdio>
#include <vector>

__global__ void Accumulate(float* a, std::size_t n, std::size_t dim, float* b) {
	int j = threadIdx.x + blockIdx.x * blockDim.x;
	if(j < dim) {
		b[j] = 0;
		for(int i = 0; i < n; ++i) b[j] += a[i * dim + j];
	}
}
