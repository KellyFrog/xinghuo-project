#include "hip/hip_runtime.h"
#include "matrix.cuh"
#include <cstdio>
#include <cstdlib>
#include <random>
#include <cassert>

int main() {
	size_t n = 1 << 9;
	size_t m = 3 << 8 | 5;
	size_t k = 5 << 7 | 9;
	float* a = (float*)std::malloc(n * m * sizeof(float));
	float* b = (float*)std::malloc(m * k * sizeof(float));
	float* c = (float*)std::malloc(n * k * sizeof(float));
	float* d = (float*)std::malloc(n * k * sizeof(float));
	std::mt19937 rnd(0x114514);
	for(int i = 0; i < n; ++i) {
		for(int j = 0; j < m; ++j) {
			a[i * m + j] = rnd() / 1e7;
			//a[i * m + j] = (i == j);
		}
	}
	for(int i = 0; i < m; ++i) {
		for(int j = 0; j < k; ++j) {
			b[i * k + j] = rnd() / 1e7;
			//b[i * k + j] = (i == j);
		}
	}
	float* device_a = NULL;
	float* device_b = NULL;
	float* device_c = NULL;
	assert(hipMalloc((void**) &device_a, n * m * sizeof(float)) == hipSuccess);
	assert(hipMalloc((void**) &device_b, m * k * sizeof(float)) == hipSuccess);
	assert(hipMalloc((void**) &device_c, n * k * sizeof(float)) == hipSuccess);
	assert(hipMemcpy(device_a, a, n * m * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
	assert(hipMemcpy(device_b, b, m * k * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
	for(int t = 0; t < (1 << 5); ++t) {
		dim3 block(BSIZE, BSIZE);
		dim3 numBlock(n / BSIZE + 1, k / BSIZE + 1);
		MatrixMul<<<numBlock, block>>>(device_a, device_b, n, m, k, device_c);
	}
	for(int i = 0; i < n; ++i) {
		for(int j = 0; j < k; ++j) {
			float res = 0;
			for(int p = 0; p < m; ++p) {
				res += a[i * m + p] * b[p * k + j];
			}
			c[i * k + j] = res;
		}
	}
	assert(hipMemcpy(d, device_c, n * k * sizeof(float), hipMemcpyDeviceToHost) == hipSuccess);
	for(int i = 0; i < n; ++i) {
		for(int j = 0; j < k; ++j) {
			assert(fabs(d[i * k + j] - c[i * k + j]) / std::max(1.0f, fabs(d[i * k + j])) < 1e-3);
		}
	}
	assert(hipGetLastError() == hipSuccess);
	assert(hipFree(device_a) == hipSuccess);
	assert(hipFree(device_b) == hipSuccess);
	assert(hipFree(device_c) == hipSuccess);
	std::free(a);
	std::free(b);
	std::free(c);

	return 0;
}
