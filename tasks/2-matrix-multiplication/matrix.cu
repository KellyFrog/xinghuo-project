#include "hip/hip_runtime.h"
#include "matrix.cuh"
#include <cstdio>

__global__ void MatrixMul(const float* a, const float* b, std::size_t n, std::size_t m, std::size_t k, float* c) {
	__shared__ float sa[BSIZE][BSIZE];
	__shared__ float sb[BSIZE][BSIZE];
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int x = threadIdx.x;
	int y = threadIdx.y;
	float res = 0;
	for(int p = 0; p < m; p += BSIZE) {
		if(y == 0) {
			for(int q = 0; q < BSIZE; ++q) {
				if(i < n && p + q < m) sa[x][q] = a[i * m + p + q];
				else sa[x][q] = 0;
			}
		}
		if(x == 0) {
			for(int q = 0; q < BSIZE; ++q) {
				if(p + q < m && j < k) {
					sb[q][y] = b[(p + q) * k + j];
				}
				else sb[q][y] = 0;
			}
		}
		__syncthreads();
		for(int q = 0; q < BSIZE; ++q) res += sa[x][q] * sb[q][y];
		__syncthreads();
	}
	if(i < n && j < k) c[i * k + j] = res;
}
